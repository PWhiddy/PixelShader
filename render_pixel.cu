#include "hip/hip_runtime.h"
//#include "noise.h"
#include "cuda_noise.h"
#include "cutil_math.h"

__device__ float2 rotate(float2 p, float a)
{
    return make_float2(p.x*cos(a) - p.y*sin(a),
                       p.y*cos(a) + p.x*sin(a));
}

__device__ float sdSphere(float3 p, float r) {
    return length(p)-r;
}

__device__ float sdBox( float3 p, float3 b )
{
      float3 d = fabs(p) - b;
      return fminf(fmaxf(d.x,fmaxf(d.y,d.z)),0.0f) + length(fmaxf(d,make_float3(0.0f)));
}

__device__ float fractalNoise(float3 p) {
    p += 300.0f;
    float result = 0.0f;
    result += rng::simplexNoise(p*1.0, 1.0, 123) * 1.0f;
    result += rng::simplexNoise(p*2.0, 1.0, 123) * 0.5f;
    result += rng::simplexNoise(p*4.0, 1.0, 123) * 0.25f;
    result += rng::simplexNoise(p*8.0, 1.0, 123) * 0.125f;
    result += rng::simplexNoise(p*16.0, 1.0, 123) * 0.0625f;
    result += rng::simplexNoise(p*32.0, 1.0, 123) * 0.03125f;
    result += rng::simplexNoise(p*64.0, 1.0, 123) * 0.015625f;
    return result;
}

__device__ float map(float3 p) {
    float d;
    d =  sdSphere(p, 0.7)/*+(sin(38.0*p.x)+sin(47.0*p.y)+sin(21.0*p.z))*0.1*/+0.1*fractalNoise(p*2.5);
    d = fminf(-sdBox(p, make_float3(2.0,2.0,2.0)), d);
    return d;
}

__device__ float3 calcNormal( float3 pos )
{
    float2 e = make_float2(1.0,-1.0)*0.5773*0.0005;
    return normalize( make_float3(e.x,e.y,e.y)*map( pos + make_float3(e.x,e.y,e.y) ) + 
					  make_float3(e.y,e.y,e.x)*map( pos + make_float3(e.y,e.y,e.x) ) + 
					  make_float3(e.y,e.x,e.y)*map( pos + make_float3(e.y,e.x,e.y) ) + 
					  make_float3(e.x,e.x,e.x)*map( pos + make_float3(e.x,e.x,e.x) ) );
}

__global__ void render_pixel ( 
    uint8_t *image, 
    int x_dim, 
    int y_dim, 
    int time_step
){
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    if (x >= x_dim || y >= y_dim) return;

    float time = float(time_step);
    
    // Create Normalized UV image coordinates
    float uvx =  float(x)/float(x_dim)-0.5;
    float uvy = -float(y)/float(y_dim)+0.5;
    uvx *= float(x_dim)/float(y_dim);     

    float3 light_dir = normalize(make_float3(0.1, 1.0, -0.5));

    // Set up ray originating from camera
    float3 ray_pos = make_float3(0.0, 0.0, -1.5);
    float2 pos_rot = rotate(make_float2(ray_pos.x, ray_pos.z), 0.0);
    ray_pos.x = pos_rot.x;
    ray_pos.z = pos_rot.y;
    float3 ray_dir = normalize(make_float3(uvx,uvy,0.5));
    float2 dir_rot = rotate(make_float2(ray_dir.x, ray_dir.z), 0.0);
    ray_dir.x = dir_rot.x;
    ray_dir.z = dir_rot.y;

    for (int i=0; i<1024; i++) {
        float dist = map(ray_pos);
        if (dist < 0.002 || dist > 100.0) break;
        ray_pos += dist * ray_dir * 0.15;
    }

    float3 background = make_float3(0.87);
    float3 normal = calcNormal(ray_pos);
    float value = dot(normal,light_dir);
    float3 color = make_float3(value, value, value);
    if (length(ray_pos) > 10.0) color = background;
    //color = make_float3(rng::simplexNoise(make_float3(uvx*25.0,uvy*25.0,0.0)+100.0, 1.0, 123));

    /*
    const float3 dir_to_light = normalize(light_dir);
    const float occ_thresh = 0.001;
    float d_accum = 1.0;
    float light_accum = 0.0;
    float temp_accum = 0.0;
    

    // Trace ray through volume
    for (int step=0; step<512; step++) {
        // At each step, cast occlusion ray towards light source
        float c_density = get_cellF(ray_pos, vd, volume);
        float3 occ_pos = ray_pos;
        ray_pos += ray_dir*step_size;
        // Don't bother with occlusion ray if theres nothing there
        if (c_density < occ_thresh) continue;
        float transparency = 1.0;
        for (int occ=0; occ<512; occ++) {
            transparency *= fmax(1.0-get_cellF(occ_pos, vd, volume),0.0);
            if (transparency < occ_thresh) break;
            occ_pos += dir_to_light*step_size;
        }
        d_accum *= fmax(1.0-c_density,0.0);
        light_accum += d_accum*c_density*transparency;
        if (d_accum < occ_thresh) break;
    }
    
    // gamma correction
    light_accum = pow(light_accum, 0.45);
    */
    const int pixel = 3*(y*x_dim+x);
    image[pixel+0] = (uint8_t)(fmin(255.0*color.x, 255.0));
    image[pixel+1] = (uint8_t)(fmin(255.0*color.y, 255.0));
    image[pixel+2] = (uint8_t)(fmin(255.0*color.z, 255.0));
}
